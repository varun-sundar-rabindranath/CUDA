/*
 * Cuda vector addition code.
 */

#include <iostream>
#include <cstdlib>
#include <cassert>
using namespace std;

#include <hip/hip_runtime.h> // CUDA include

__global__ void vecadd(const float* A, const float* B, float* C, int length) {

  int id = blockDim.x * blockIdx.x + threadIdx.x;

  if (id < length)
    C[id] = A[id] + B[id];
}

int main(int argc, char* argv[]) {

  if (argc != 2) {
    cerr<<"Too many or too few arguments - Usage : vecadd <size of vectors>"<<endl;
  }

  int vec_size = atoi(argv[1]);
  assert(vec_size > 0 && "Invalid input size");

  int data_size = vec_size * sizeof(float);

  float* A = NULL;
  float* B = NULL;
  float* C = NULL;

  /* Allocate cpu memory */
  A = (float*) malloc(data_size);
  B = (float*) malloc(data_size);
  C = (float*) malloc(data_size);
  assert(A != NULL && B != NULL && C != NULL && "Cannot alloc memory");

  /* Fill inputs :
   * if vec_size is 10, then fill A and B as follows,
   * A : 0 1 2 3 4 5 6 7 8 9
   * B : 9 8 7 6 5 4 3 2 1 0
   */
  for (int vec_iter = 0; vec_iter < vec_size; vec_iter++) {
    A[vec_iter] = vec_iter;
    B[vec_iter] = vec_size - vec_iter - 1;
  }

  float* gpuA = NULL;
  float* gpuB = NULL;
  float* gpuC = NULL;

  hipError_t err;

  /* Allocate GPU memory */
  err = hipMalloc(&gpuA, data_size);
  assert(err == hipSuccess && "hipMalloc fail");
  err = hipMalloc(&gpuB, data_size);
  assert(err == hipSuccess && "hipMalloc fail");
  err = hipMalloc(&gpuC, data_size);
  assert(err == hipSuccess && "hipMalloc fail");

  /* Load inputs to gpu memory */
  err = hipMemcpy(gpuA, A, data_size, hipMemcpyHostToDevice);
  assert(err == hipSuccess && "hipMemcpy fail");
  err = hipMemcpy(gpuB, B, data_size, hipMemcpyHostToDevice);
  assert(err == hipSuccess && "hipMemcpy fail");

  /* Execute function on GPU */
  dim3 block_dim(512, 1, 1);
  dim3 grid_dim((vec_size / 512) + 1, 1, 1);

  vecadd<<<grid_dim, block_dim>>>(gpuA, gpuB, gpuC, vec_size);

  hipDeviceSynchronize(); // Wait till gpu completes execution

  /* Copy results back to CPU */
  err = hipMemcpy(C, gpuC, data_size, hipMemcpyDeviceToHost);
  assert(err == hipSuccess && "hipMemcpy fail");

  /* Check results */
  /* All elements of C should be vec_size - 1 */
  bool vec_add_pass = true;
  for (int vec_iter = 0; vec_iter < vec_size; vec_iter++) {
    if(C[vec_iter] != vec_size - 1) { vec_add_pass = false; break; }
  }
  if (vec_add_pass) { cout<<"Vector addition pass"<<endl; }
              else  { cout<<"Vector addition fail"<<endl; }

  /* Free cpu memory */
  free(A);
  free(B);
  free(C);

  /* Free cuda memory */
  hipFree(gpuA);
  hipFree(gpuB);
  hipFree(gpuC);

  return 0;
}