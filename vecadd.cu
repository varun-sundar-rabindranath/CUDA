/*
 * Cuda vector addition code.
 */

#include <iostream>
#include <cstdlib>
#include <cassert>
using namespace std;

#include <hip/hip_runtime.h> // CUDA include

/* Kernel function */
__global__ void vecadd(const float* A, const float* B, float* C) {

  int id = blockDim.x * blockIdx.x + threadIdx.x;

  C[id] = A[id] + B[id];
}

int main(int argc, char* argv[]) {

  assert(argc == 1);

  /* Lets add vectors of length 1024 */
  int vec_length = 24;

  int data_size = vec_length * sizeof(float);

  hipError_t err;

  /* CPU memory */ /* Resptve. GPU memory*/
  float* A = NULL; float* gpuA = NULL;
  float* B = NULL; float* gpuB = NULL;
  float* C = NULL; float* gpuC = NULL;

  /******************* 1. Allocate cpu memory *************************/
  A = (float*) malloc(data_size);
  B = (float*) malloc(data_size);
  C = (float*) malloc(data_size);
  assert(A != NULL && B != NULL && C != NULL && "Cannot alloc memory");

  /* Fill inputs :
   * A : 0    1    2    3    4    5    6 7 8 9 ...
   * B : 1023 1022 1021 1020 1019 1018 ...
   */
  for (int vec_iter = 0; vec_iter < vec_length; vec_iter++) {
    A[vec_iter] = vec_iter;
    B[vec_iter] = vec_length - vec_iter - 1;
  }

  /******************* 2. Allocate GPU memory ************************/
  err = hipMalloc(&gpuA, data_size);
  assert(err == hipSuccess && "hipMalloc fail");
  err = hipMalloc(&gpuB, data_size);
  assert(err == hipSuccess && "hipMalloc fail");
  err = hipMalloc(&gpuC, data_size);
  assert(err == hipSuccess && "hipMalloc fail");

  /******************* 3. Load inputs to GPU memory ******************/
  err = hipMemcpy(gpuA, A, data_size, hipMemcpyHostToDevice);
  assert(err == hipSuccess && "hipMemcpy fail");
  err = hipMemcpy(gpuB, B, data_size, hipMemcpyHostToDevice);
  assert(err == hipSuccess && "hipMemcpy fail");

  /******************* 4. Execute function on GPU ********************/
  /* Execute function on GPU */
  dim3 block_dim(8, 1, 1);
  dim3 grid_dim(3, 1, 1);

  vecadd<<<grid_dim, block_dim>>>(gpuA, gpuB, gpuC);
  hipDeviceSynchronize(); // Wait till gpu completes execution

  /****************** 5. Copy results back to CPU ********************/
  err = hipMemcpy(C, gpuC, data_size, hipMemcpyDeviceToHost);
  assert(err == hipSuccess && "hipMemcpy fail");

  /* Check results */
  /* All elements of C should be 1023 */
  bool vec_add_pass = true;
  for (int vec_iter = 0; vec_iter < vec_length; vec_iter++) {
    if(C[vec_iter] != vec_length - 1) { vec_add_pass = false; break; }
  }
  if (vec_add_pass) { cout<<"Vector addition pass"<<endl; }
              else  { cout<<"Vector addition fail"<<endl; }

  /* Free cpu memory */
  free(A);
  free(B);
  free(C);

  /* Free cuda memory */
  hipFree(gpuA);
  hipFree(gpuB);
  hipFree(gpuC);

  return 0;
}
